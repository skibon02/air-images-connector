#include "hip/hip_runtime.h"
#include <opencv4/opencv2/opencv.hpp>
#include <chrono>
#include <iostream>

#include <vector>
#define GLM_FORCE_RADIANS
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>

using namespace cv;
using namespace std;
using namespace chrono;

const float RATIO = 6.0/4.0;

struct Point{
    float x;
    float y;
};

struct PhotoCamera {
    Mat img;
    Mat img_warped;
    float RASKOLBAS = 14.4298679;

    glm::vec3 pos;
    std::vector<float> angles;
    std::vector<float> angles_offs;

    std::vector<glm::vec3> axis;
    glm::vec3 dirLine;
    std::vector<glm::vec3> dirRays;
    float perc;
    std::vector<glm::vec3> dots;

    string filename;

    PhotoCamera(glm::vec3 pos, std::vector<float> angles, string filename) : angles_offs(std::vector<float>(3)) {
        img = imread(filename);
        angles_offs = {-1.2, -2.4, 0};
        // angles_offs = {-2, -6, 1}; 
        this->perc = 1;
        this->angles = angles;
        this->pos = pos;    
        this->filename = filename;

        axis.resize(3);
        dirRays.resize(4);
        dots.resize(4);
        
        Update();
    }

    void Update() {
        angles[0] = 0;
        angles[1] = 0; 

        axis[0] = glm::vec3(1.0f, 0.0f, 0.0f);
        axis[1] = glm::vec3(0.0f, 1.0f, 0.0f);
        axis[2] = glm::vec3(0.0f, 0.0f, 1.0f);

        //axis
        //0 - по направлению
        //1 - вверх
        //2 - вправо
        auto mat_rot = glm::rotate(glm::mat4(1.0f), glm::radians(-angles[2] - angles_offs[2]), axis[1]);
        auto mat_rot3 = glm::mat3(mat_rot);
        axis[0] = mat_rot3 * axis[0];
        axis[2] = mat_rot3 * axis[2];

        mat_rot = glm::rotate(glm::mat4(1.0f), glm::radians(angles[1] + angles_offs[1]), axis[2]);
        mat_rot3 = glm::mat3(mat_rot);
        axis[0] = mat_rot3 * axis[0];
        axis[1] = mat_rot3 * axis[1];

        mat_rot = glm::rotate(glm::mat4(1.0f), glm::radians(-angles[0] - angles_offs[0]), axis[0]);
        mat_rot3 = glm::mat3(mat_rot);
        axis[2] = mat_rot3 * axis[2];
        axis[1] = mat_rot3 * axis[1];


        dirLine = axis[1] * -3.0f;

        dirRays[0] = glm::mat3(glm::rotate(glm::rotate(glm::mat4(1.0f), glm::radians(RASKOLBAS), axis[2]), glm::radians(RASKOLBAS*RATIO), axis[0]))*dirLine * 500.0f;
        dirRays[1] = glm::mat3(glm::rotate(glm::rotate(glm::mat4(1.0f), glm::radians(RASKOLBAS), axis[2]), glm::radians(-RASKOLBAS*RATIO), axis[0]))*dirLine * 500.0f;
        dirRays[2] = glm::mat3(glm::rotate(glm::rotate(glm::mat4(1.0f), glm::radians(-RASKOLBAS), axis[2]), glm::radians(-RASKOLBAS*RATIO), axis[0]))*dirLine * 500.0f;
        dirRays[3] = glm::mat3(glm::rotate(glm::rotate(glm::mat4(1.0f), glm::radians(-RASKOLBAS), axis[2]), glm::radians(RASKOLBAS*RATIO), axis[0]))*dirLine * 500.0f;

        for(int i = 0; i < 4; i++) {
            dots[i]  = glm::normalize(dirRays[i]);
            dots[i] = (-pos.y/dots[i].y)*dots[i]*perc + pos;
        }


        Point2f src_vertices[4];

        Point2f dst_vertices[4];
        for(int i = 0; i < 4; i++) {
            dst_vertices[i].y = dots[i].x + 800;
            dst_vertices[i].x = dots[i].z + 300;

        }
        src_vertices[0] = {0, 0};
        src_vertices[1] = {6000, 0};
        src_vertices[2] = {6000, 4000};
        src_vertices[3] = {0, 4000};
        auto dotsTransf = getPerspectiveTransform(src_vertices, dst_vertices, cv::INTER_LINEAR);

        warpPerspective(img, img_warped, dotsTransf, {900, 900});

        // imshow("img", img);
        // imshow("warped", img_warped);
        // waitKey(); 
    }


};

// __global__ void dothing(char* mem1, char* mem2) {
//     int i = threadIdx.x + blockIdx.x * 1024;
//     for(int j = 0; j < 1000; j++) {
//         mem1[i] = 0;
//         mem2[i] = 255;
//     }
// }
// #define THREADS_PER_BLOCK 1024
int main() {

    vector<PhotoCamera> photoCameras;

    const float COEF_X = 110480;
    const float COEF_Y = 55654;
    std::vector<std::vector<float>> angles = {
        {-05.45, -10.45, -179.58}, {-03.43, -09.27, -177.48}, {-05.87, -12.14, 178.23}, 

        {-10.10, -11.78, 177.98},
        {-06.85, -13.04, 179.90},
        {-09.62, -11.91, -179.39},
        {-10.52, -13.17, -178.86},
        {-06.78, -14.10, -176.86},
    {-06.77, -12.18, -177.51},
    {-06.05, -12.59, -177.49},
    {-07.16, -12.77, -177.74},
    {-07.15, -13.74, -179.07},
    {-06.11, -17.00, -175.62},
    {-08.74, -10.15, -179.42},
    {-07.65, -12.08, -178.54},
    {-05.98, -11.52, 178.64},
    {-06.56, -11.31, 178.03},
    {-05.70, -07.43, 175.38},
    {-04.92, -09.45, -179.05},
    {-07.67, -12.80, -178.61},
    {02.87, -00.38, -02.62}, {04.68, -02.56, -00.02}, {04.10, -01.98, -03.26}, {04.98, -02.82, -03.45}, {06.11, -04.27, -03.68}, 
    {07.08, -04.76, -04.11}, {06.15, -05.47, -02.01}, {06.10, -06.80, -04.98}, {05.06, -06.50, -05.33}, {06.19, -07.60, -08.18}, 
    {05.97, -05.20, -02.11}, {04.75, -04.47, 00.08}, {06.51, -09.35, -05.98}, {06.62, -08.23, 01.21}, {06.81, -07.84, 01.67},
    {04.79, -08.53, -00.83},
    };
    std::vector<::Point> gps_coords = {
        {59.84265850, 31.47160150}, {59.84228240, 31.47159460 }, {59.84190420, 31.47159500},
        
        {59.84153520, 31.47161250},
        {59.84115850, 31.47162660},
        {59.84078280, 31.47163910},	
        {59.84040940, 31.47162940},
        {59.84003730, 31.47161260},
        {59.83965250, 31.47160240},
        {59.83927620, 31.47159090},
        {59.83889160, 31.47158680},
        {59.83851880, 31.47160170},
        {59.83814570, 31.47159980},
        {59.83777020, 31.47160300},
        {59.83739830, 31.47161090},
        {59.83702240, 31.47162450},
        {59.83664390, 31.47161670},
        {59.83626190, 31.47162380},
        {59.83588520, 31.47160890},
        {59.83551610, 31.47160450},
        //25..40
        {59.83535810, 31.47351560}, {59.83572970, 31.47349250}, {59.83611240, 31.47347330}, {59.83649310, 31.47347010},
        {59.83687230, 31.47347420}, {59.83724990, 31.47347520}, {59.83763040, 31.47347290}, {59.83801190, 31.47347450}, 
        {59.83838830, 31.47347030}, {59.83876860, 31.47347330}, {59.83914780, 31.47347090}, {59.83952690, 31.47346510},
        {59.83990310, 31.47347460}, {59.84027640, 31.47347250}, {59.84065270, 31.47347150}, {59.84103100, 31.47347000},
    };
    std::vector<float> altitudes = {
        273.273, 272.246, 275.501, 
        273.532, 272.589, 271.905, 273.361, 270.790,
        276.101, 275.673, 274.732, 272.845, 273.273, 273.701, 273.532, 273.873, 273.189, 272.161, 272.845,274.473 ,272.333, 273.617,
        //25..40
        275.673, 276.101, 275.416, 277.044, 276.872, 276.272, 277.044, 276.272, 276.016,
        276.959, 273.445, 275.416, 274.045, 276.360, 276.016, 277.216,
    };
    ::Point fir_gps = gps_coords[0];
    int i = 0;
    int img_id = 5;

    Mat merged;
    for(auto& coord : gps_coords) {
        coord.x -= fir_gps.x;
        coord.y -= fir_gps.y;
        coord.x *= COEF_X;
        coord.y *= COEF_Y;


        auto cam_pos = glm::vec3(coord.x, altitudes[i], coord.y);

        PhotoCamera cam(cam_pos, angles[i], "images/" + to_string(img_id) + ".JPG");
        cout << cam.dots[0].x << " ";
        cout << cam.dots[1].x << " ";
        cout << cam.dots[2].x << " ";
        cout << cam.dots[3].x << " ";
        cout << endl;
        photoCameras.push_back(cam);

        if(merged.empty()) {
            merged = photoCameras[i].img_warped;
        }
        else {
            unsigned char* p_start = (unsigned char*)photoCameras[i].img_warped.datastart;
            unsigned char* mrg_start = (unsigned char*)merged.datastart;
            for(int j = 0; j < photoCameras[i].img_warped.cols * photoCameras[i].img_warped.rows; j++) {
                if(p_start[0] > 0 && p_start[1] > 0 && p_start[2] > 0) {
                    mrg_start[0] = p_start[0];
                    mrg_start[1] = p_start[1];
                    mrg_start[2] = p_start[2];
                }
                p_start += photoCameras[i].img_warped.elemSize();
                mrg_start += photoCameras[i].img_warped.elemSize();
            }
        }

        i++;
        img_id++;
    }
    imshow("merged", merged);
    waitKey();
    while(true) {
        bool isFirst = true;
        i = 0;
        for(auto &photoCam : photoCameras) {
            photoCam.Update();

            //merge
            if(isFirst) {
                merged = photoCameras[i].img_warped;
                isFirst = false;
            }
            else {
                unsigned char* p_start = (unsigned char*)photoCameras[i].img_warped.datastart;
                unsigned char* mrg_start = (unsigned char*)merged.datastart;
                for(int j = 0; j < photoCameras[i].img_warped.cols * photoCameras[i].img_warped.rows; j++) {
                    if(p_start[0] > 0 && p_start[1] > 0 && p_start[2] > 0) {
                        mrg_start[0] = p_start[0];
                        mrg_start[1] = p_start[1];
                        mrg_start[2] = p_start[2];
                    }
                    p_start += photoCameras[i].img_warped.elemSize();
                    mrg_start += photoCameras[i].img_warped.elemSize();
                }
            }
            i++;
        }

        imshow("merged", merged);
        auto c = waitKey();
        switch (c) {
            case 'o':
    
                for(auto &photoCam : photoCameras) {
                    photoCam.angles_offs[1]+=0.3;
                    cout << "pitch offs: " << photoCam.angles_offs[1] << endl;
                }
                break;
            case 'p':
    
                for(auto &photoCam : photoCameras) {
                    photoCam.angles_offs[1]-=0.3;
                    cout << "pitch offs: " << photoCam.angles_offs[1] << endl;
                }
    
                break;


        case 'u':

            for(auto &photoCam : photoCameras) {
                photoCam.angles_offs[0]+=0.3;
                cout << "roll offs: " << photoCam.angles_offs[0] << endl;
            }
            break;
        case 'i':

            for(auto &photoCam : photoCameras) {
                photoCam.angles_offs[0]-=0.3;
                cout << "roll offs: " << photoCam.angles_offs[0] << endl;
            }

            break;

        case 'k':

            for(auto &photoCam : photoCameras) {
                photoCam.angles_offs[2]+=0.3;
                cout << "yaw offs: " << photoCam.angles_offs[2] << endl;
            }
            break;
        case 'l':

            for(auto &photoCam : photoCameras) {
                photoCam.angles_offs[2]-=0.3;
                cout << "yaw offs: " << photoCam.angles_offs[2] << endl;
            }

        break;
        }
    }


    // Mat img1, img2;
    // img1 = imread("images/img1.jpg");
    // img2 = imread("images/img2.jpg");

    // void *cudaImg1, *cudaImg2;
    // int sz1 = img1.cols*img1.rows*img1.elemSize();
    // int sz2 = img2.cols*img2.rows*img2.elemSize();
    // cout << "size: " << sz1 << endl;
    // hipMalloc((void**)&cudaImg1, sz1);
    // hipMalloc((void**)&cudaImg2, sz2);

    // auto p1 = system_clock::now();
    // hipMemcpy(cudaImg1, img1.datastart, sz1, hipMemcpyHostToDevice);
    // hipMemcpy(cudaImg2, img2.datastart, sz2, hipMemcpyHostToDevice);

    // dothing<<<sz1/THREADS_PER_BLOCK,1024>>>((char*)cudaImg1, (char*)cudaImg2);

    // hipMemcpy((void*)img1.datastart, cudaImg1, sz1, hipMemcpyDeviceToHost);
    // hipMemcpy((void*)img2.datastart, cudaImg2, sz2, hipMemcpyDeviceToHost);
    // // char* img1data = (char*)img1.datastart;
    // // char* img2data = (char*)img2.datastart;
    // // for(int i = 0; i < sz1; i++) {
    // //     img1data[i] = 0;
    // //     img2data[i] = 255;
    // // }
    // auto p2 = system_clock::now();
    // cout << duration_cast<milliseconds>(p2-p1).count() << "ms" << endl;

    // // imshow("i1", img1);
    // // imshow("i2", img2);
    // // waitKey();


    // hipFree(cudaImg1);
    // hipFree(cudaImg2);


    return 0;
}